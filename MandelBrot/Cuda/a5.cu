#include "hip/hip_runtime.h"
//Tim McMullen Assignment 5 - 159.735 parallel, Cuda

//nvcc -o a5 a5.cu -I/home/MASSEY/06222757/Desktop/cfitsio -L/home/MASSEY/06222757/Desktop/cfitsio fitsfile.o -l cfitsio -lm

#include <iostream>
#include <cmath>
#include <ctime>
#include <string>
#include <hip/hip_runtime.h>

#include "fitsfile.h"

#define npix 2048
#define WIDTH 4.0
#define max_value 256
#define filename "output.fits"

void write_to_fits(float* h, int nrows, int ncols){

  FitsFile ff;
  ff.createFile(filename);
  ff.createImage2D(ncols, nrows, FLOAT_IMG);

  // Write to image file row by row
  for (int i = 0; i < nrows; ++i) {
    long firstel = i * ncols + 1;
    long ntotal = ncols;
    ff.writeImage(&h[i*ncols], firstel, ntotal);
  }
  ff.closeFile();

}

// Kernel that executes on the CUDA device. This is executed by ONE stream processor
__global__ void calc_pixels(float* image_out, int size, float max_val, float width){

  int i;
  float row, col, ii, jj, x1, y1, pix_size;
  // set the x and y values
  int npixx = size;
  int npixy = size;
  // What element of the array does this thread work on
  i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i > npixy*npixx)return;

  y1 = x1 = -(width/2);
  row = i/npixx;
  col = i%npixx;
  pix_size = (float)width/npixx;
  ii = x1+col*pix_size;
  jj = y1+row*pix_size;

  float count, temp, lensq, x, y;
  lensq=1e12;
  x = 0;
  y = 0;
  count = 0;
  do {
    temp = x * x - y * y + ii;
    y = 2 * x * y + jj;
    x = temp;

    lensq = x * x + y * y;
    ++count;
  } while ( (lensq < width) && (count < max_value) );

  image_out[i] = count;

}

int main(int argc, char* argv[]){

  int image_size, totalSize;
  float *Buffer, *image_out;

  totalSize = npix*npix;

  Buffer = new float[totalSize];
  image_size = totalSize*sizeof(float);
  // Allocate in DEVICE memory
  hipMalloc(&image_out, image_size);

  // calculate number of blocks per grid required to run sobel operator on image
  int threadsPerBlock = 256;
  int blocksPerGrid = (totalSize + threadsPerBlock - 1) / threadsPerBlock;

  // Invoke kernel
  calc_pixels<<<blocksPerGrid, threadsPerBlock>>>(image_out, npix, max_value, WIDTH);

  // Copy result from device memory into host memory
  hipMemcpy(Buffer, image_out, image_size, hipMemcpyDeviceToHost);

  hipFree(image_out);

  write_to_fits(Buffer, npix, npix);

  free(Buffer);
}
